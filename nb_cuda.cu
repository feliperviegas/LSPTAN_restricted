#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <cfloat>
#include <sstream>
#include "io.hpp"
#include "evaluate.h"
#include <sys/time.h>

#define CUDA_CHECK_RETURN(value) { \
               hipError_t _m_cudaStat = value;\
               if (_m_cudaStat != hipSuccess) {\
                       fprintf(stderr, "Error %s at line %d in file %s\n",\
                                       hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
                                       exit(1);\
               }}

#define SIZE_TRAIN 256
#define SIZE_CLASS 128

std::string index(int doc, int term){
	stringstream t, d;
	d << doc;
	t << term;
	return d.str() + "-" + t.str();
}

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

__global__ void trainning_kernel2(int *freqClassVector, double *matrixTermFreq,
		double* totalFreqClassVector, int *docTestIndexVector,
		int *docTestVector, double *docTestFreqVector, double *probClasse,
		int numClasses, int numTerms, int numDocsTest, double *freqTermVector,
		double totalTermFreq, int totalTerms, double lambda, double alpha,
		int numDocs, double *modeloNB) {

	int vecs, len, term;
	double freq;
	double prob, nt, maiorProb;
	extern __shared__ double temp[]; // used to hold segment of the vector (size nthreads)
	// plus 3 integers (vecs, len, partial sum) at the end
	int tid = threadIdx.x;

	if (tid == 0) {  // thread 0 calculates vecs and len
		//vecs - vector size
		temp[blockDim.x + 1] = (docTestIndexVector[blockIdx.x + 1]
				- docTestIndexVector[blockIdx.x]);
		// len - number of segments (size nthreads) of the vector
		if(temp[blockDim.x + 1] > blockDim.x)
    		temp[blockDim.x + 2] = ceil(temp[blockDim.x + 1] / (double) blockDim.x);
    	else
    		temp[blockDim.x + 2] = 1.0;
		maiorProb = -99999.9;
	}
	__syncthreads();

	vecs = temp[blockDim.x + 1]; // communicate vecs and len's values to other threads
	len = (int) temp[blockDim.x + 2];

	for (int c = 0; c < numClasses; c++) {
		if (tid == 0) {
			// partial sum initialization
			temp[blockDim.x + 3] = log((freqClassVector[c] + alpha) / (numDocs + alpha * numClasses));
		}
		__syncthreads();
		for (int b = 0; b < len; b++) { // loop through 'len' segments
			// first, each thread loads data into shared memory
			if ((b * blockDim.x + tid) >= vecs) // check if outside 'vec' boundary
				temp[tid] = 0.0;
			else {
				term = docTestVector[docTestIndexVector[blockIdx.x] + b * blockDim.x + tid];
				freq = docTestFreqVector[docTestIndexVector[blockIdx.x] + b * blockDim.x + tid];
				prob = (matrixTermFreq[c * numTerms + term] + alpha) / (totalFreqClassVector[c] + alpha * totalTerms);
				nt = freqTermVector[term] / totalTermFreq;
				prob = lambda * nt + (1.0 - lambda) * prob;
				if(freqTermVector[term] != 0){
					temp[tid] = freq * log(prob);
		        }
		        else{
		          temp[tid] = 0.0;
		        }
			}
			__syncthreads();

			// next, perform binary tree reduction on shared memory
			for (int d = blockDim.x >> 1; d > 0; d >>= 1) {
				if (tid < d)
					temp[tid] += (tid + d) >= vecs ? 0.0 : temp[tid + d];
				__syncthreads();
			}

			// first thread puts partial result into shared memory
			if (tid == 0) {
				temp[blockDim.x + 3] += temp[0];
			}
			__syncthreads();
		}
		// finally, first thread puts result into global memory
		if (tid == 0) {
			modeloNB[blockIdx.x * numClasses + c] = temp[blockDim.x + 3];
			if (c == 0) {
				maiorProb = temp[blockDim.x + 3];
			} else if (temp[blockDim.x + 3] > maiorProb) {
				maiorProb = temp[blockDim.x + 3];
			}
		}
		__syncthreads();
	}

	if (tid == 0) {
		probClasse[blockIdx.x] = maiorProb;
	}
}

__global__ void super_parent_freq(int *docIndexVector, int *docVector,
		double *docFreqVector, int *docClassVector, double *totalTermClassSp,
		int numTerms, int numDocs, int totalTerms, int numClasses) {

	int sp = blockIdx.x * blockDim.x + threadIdx.x;
	int term;
	double freq;

	if (sp < numTerms) {
		for (int c = 0; c < numClasses; c++)
			totalTermClassSp[c * numTerms + sp] = 0.0;
		for (int d = 0; d < numDocs; d++) {
			int clas = docClassVector[d];
			int inicio = docIndexVector[d];
			int fim = docIndexVector[d + 1];

			for (int t = inicio; t < fim; t++) {
				term = docVector[t];
				freq = docFreqVector[t];
				if (term == sp && freq > 0) {
					for (int t2 = inicio; t2 < fim; t2++) {
						term = docVector[t2];
						freq = docFreqVector[t2];
						if (term != sp){
							totalTermClassSp[clas * numTerms + sp] += freq;
						}
					}
				}
			}
		}
	}
}

__global__ void find_sp_kernel(int *docIndexVector,
		int *docVector, double *docFreqVector, int numClasses,
		int numTerms, int numDocs, int totalTerms, int *hasSp, int sp) {

	int vecs, len, term;
	double freq;
	__shared__ int aux[2]; // used to hold segment of the vector (size nthreads)
	// plus 3 integers (vecs, len, partial sum) at the end
	int tid = threadIdx.x;

	if (tid == 0) {  // thread 0 calculates vecs and len
		//vecs - vector size
		aux[0] = (docIndexVector[blockIdx.x + 1] - docIndexVector[blockIdx.x]);
		// len - number of segments (size nthreads) of the vector
		if(aux[0] > blockDim.x)
			aux[1] = ceil(aux[0] / (double) blockDim.x);
		else
			aux[1] = 1.0;
		hasSp[blockIdx.x] = 0;
	}
	__syncthreads();

	vecs = aux[0]; // communicate vecs and len's values to other threads
	len = aux[1];

	for (int b = 0; b < len; b++) { // loop through 'len' segments
		// first, each thread loads data into shared memory
		if ((b * blockDim.x + tid) < vecs){ // check if outside 'vec' boundary
			term = docVector[docIndexVector[blockIdx.x] + b * blockDim.x + tid];
			freq = docFreqVector[docIndexVector[blockIdx.x] + b * blockDim.x + tid];
			if(term == sp && freq > 0){
				hasSp[blockIdx.x] = 1;
			}
		}
		__syncthreads();
	}
}

__global__ void init_tableProb(int numTerms, int numClasses,double* probSp) {

	int termId = blockIdx.x * blockDim.x + threadIdx.x;
	int i;

	if (termId < numTerms) {
		for (i = 0; i < numClasses; i++) {
			probSp[i * numTerms + termId] = 0.0;
		}
	}
}


__global__ void compute_frequency(int *docIndexVector, int *docVector, double *docFreqVector, 
	int *docClassVector, int numClasses, int numTerms, int numDocs, int totalTerms, int *hasSp, 
	int sp, double *probSp) {

	int vecs, len, term, clas;
	double freq;
	__shared__ int aux[2]; // used to hold segment of the vector (size nthreads)
	// plus 3 integers (vecs, len, partial sum) at the end
	int tid = threadIdx.x;
	if (tid == 0) {  // thread 0 calculates vecs and len
		//vecs - vector size
		aux[0] = (docIndexVector[blockIdx.x + 1] - docIndexVector[blockIdx.x]);
		// len - number of segments (size nthreads) of the vector
		if(aux[0] > blockDim.x)
			aux[1] = ceil(aux[0] / (double) blockDim.x);
		else
			aux[1] = 1.0;
		
	}
	__syncthreads();

	vecs = aux[0]; // communicate vecs and len's values to other threads
	len = aux[1];
	clas = docClassVector[blockIdx.x];

	if(hasSp[blockIdx.x] == 1){
		for (int b = 0; b < len; b++) { // loop through 'len' segments
			// first, each thread loads data into shared memory
			if ((b * blockDim.x + tid) < vecs){ // check if outside 'vec' boundary
				term = docVector[docIndexVector[blockIdx.x] + b * blockDim.x + tid];
				freq = docFreqVector[docIndexVector[blockIdx.x] + b * blockDim.x + tid];
				atomicAdd(&(probSp[clas * numTerms + term]), freq);
			}
			__syncthreads();
		}
	}
}

__global__ void super_parent_train(int *docIndexVector, int *docVector,
		double *docFreqVector, int *docClassVector, double *totalTermClassSp,
		int numTerms, int numDocs, int totalTerms, int numClasses,
		double* probSp, int sp, double alpha) {

	int termId = blockIdx.x * blockDim.x + threadIdx.x;
	int i;

	if (termId < numTerms) {	
		for (i = 0; i < numClasses; i++) {
			probSp[i * numTerms + termId] = (probSp[i * numTerms + termId] + alpha)	/ (totalTermClassSp[i * numTerms + sp] + alpha * (double) totalTerms);
		}
	}
}



// __global__ void super_parent_train(int *docIndexVector, int *docVector,
// 		double *docFreqVector, int *docClassVector, double *totalTermClassSp,
// 		int numTerms, int numDocs, int totalTerms, int numClasses,
// 		double* probSp, int sp, double alpha, int *hasSp) {

// 	int termId = blockIdx.x * blockDim.x + threadIdx.x;
// 	int i, d, t;
// 	int term;
// 	double freq;

// 	if (termId < numTerms) {
// 		for (i = 0; i < numClasses; i++) {
// 			probSp[i * numTerms + termId] = 0;
// 		}

// 		//Calculo da Frequencia de um termo dado Super pai e a Classe
// 		for (d = 0; d < numDocs; d++) {
// 			if(hasSp[d] == 1){
// 				int clas = docClassVector[d];
// 				int inicio = docIndexVector[d];
// 				int fim = docIndexVector[d + 1];

// 				//Procurando Super Pai no documento
// 				for (t = inicio; t < fim; t++) {
// 					term = docVector[t];
// 					freq = docFreqVector[t];
// 					if (term == termId && freq > 0 && sp != termId) {
// 						// probSp[clas * numTerms + termId] += freq;
// 						atomicAdd(&(probSp[clas * numTerms + termId]), freq);
// 					}
// 				}
// 			}
// 		}
    	
// 		for (i = 0; i < numClasses; i++) {
// 			probSp[i * numTerms + termId] = (probSp[i * numTerms + termId] + alpha)	/ (totalTermClassSp[i * numTerms + sp] + alpha * (double) totalTerms);
// 		}
// 	}
// }


__global__ void super_parent_predict2(double *matrixTermFreq,
		double* totalFreqClassVector, int *docTestIndexVector,
		int *docTestVector, double *docTestFreqVector, double *probClassSp,
		int numClasses, int numTerms, int numDocsTest, double *freqTermVector,
		double totalTermFreq, int totalTerms, double lambda, double alpha,
		int sp, double *modeloNB, double *probSp, int *docClassSp, 
		int *freqClassVector, int numDocs) {

	int vecs, len, term;
	double freq;
	double prob, nt, maiorProb;
	int bestClass;
	extern __shared__ double temp[]; // used to hold segment of the vector (size nthreads)
	// plus 3 integers (vecs, len, partial sum) at the end
	int tid = threadIdx.x;

	if (tid == 0) {  // thread 0 calculates vecs and len
		//vecs - vector size
		temp[blockDim.x + 1] = (docTestIndexVector[blockIdx.x + 1] - docTestIndexVector[blockIdx.x]);
		// len - number of segments (size nthreads) of the vector
		if(temp[blockDim.x + 1] > blockDim.x)
    		temp[blockDim.x + 2] = ceil(temp[blockDim.x + 1] / (double) blockDim.x);
    	else
    		temp[blockDim.x + 2] = 1.0;
	}
	__syncthreads();

	vecs = temp[blockDim.x + 1]; // communicate vecs and len's values to other threads
	len = (int) temp[blockDim.x + 2];
	for (int c = 0; c < numClasses; c++) {
		// //-------------------------MODELO ORIGINAL-----------------------------------------
		if(tid == 0){
			temp[blockDim.x + 3] = log((freqClassVector[c] + alpha) / (numDocs + alpha * numClasses));
		}
		__syncthreads();
		for (int b = 0; b < len; b++) { // loop through 'len' segments			
			if ((b * blockDim.x + tid) >= vecs) // check if outside 'vec' boundary
				temp[tid] = 0.0;
			else {
				term = docTestVector[docTestIndexVector[blockIdx.x]	+ b * blockDim.x + tid];
				freq = docTestFreqVector[docTestIndexVector[blockIdx.x]	+ b * blockDim.x + tid];
				nt = freqTermVector[term] / totalTermFreq;
				prob = (matrixTermFreq[c * numTerms + term] + alpha) / (totalFreqClassVector[c] + alpha * totalTerms);
				if((term != sp) && (probSp[c * numTerms + term] > prob)){
				// if((term != sp)  && (probSp[c * numTerms + term] > (alpha / alpha * (double)totalTerms))){
					prob = log(lambda*nt + (1.0 - lambda)*probSp[c * numTerms + term]);
				 }
				else{
					prob = (matrixTermFreq[c * numTerms + term] + alpha) / (totalFreqClassVector[c] + alpha * totalTerms);
		 			prob = log(lambda * nt + (1.0 - lambda) * prob);	
				}
				if(freqTermVector[term] != 0){
					temp[tid] = freq * prob;
		        }
		        else{
		          temp[tid] = 0.0;
		        }
			}
		//-------------------------------------------------------------------------------
		//----------------------MODELO SIMPLIFICADO--------------------------------------
		// if(tid == 0){
		// 	temp[blockDim.x + 3] = modeloNB[blockIdx.x * numClasses + c];
		// }
		// __syncthreads();
		// for (int b = 0; b < len; b++) { // loop through 'len' segments			
		// 	if ((b * blockDim.x + tid) >= vecs) // check if outside 'vec' boundary
		// 		temp[tid] = 0.0;
		// 	else {
		// 		term = docTestVector[docTestIndexVector[blockIdx.x] + b * blockDim.x + tid];
		// 		freq = docTestFreqVector[docTestIndexVector[blockIdx.x] + b * blockDim.x + tid];
		// 		prob = (matrixTermFreq[c * numTerms + term] + alpha) / (totalFreqClassVector[c] + alpha * totalTerms);
		// 		nt = freqTermVector[term] / totalTermFreq;
		// 		prob = log(lambda * nt + (1.0 - lambda) * prob);
		// 		prob = log(lambda*nt + (1.0 - lambda)*probSp[c * numTerms + term]) - prob;	
		// 		if(freqTermVector[term] != 0 && sp != term){
		// 			temp[tid] = freq * prob;
		//         }
		//         else{
		//           temp[tid] = 0.0;
		//         }
		// 	}
		//---------------------------------------------------------------------------------
			__syncthreads();
			// next, perform binary tree reduction on shared memory
			for (int d = blockDim.x >> 1; d > 0; d >>= 1) {
				if (tid < d)
					temp[tid] += (tid + d) >= vecs ? 0.0 : temp[tid + d];
				__syncthreads();
			}
			// first thread puts partial result into shared memory
			if (tid == 0) {
				temp[blockDim.x + 3] += temp[0];
			}
			__syncthreads();
		}
		// finally, first thread puts result into global memory
		if (tid == 0) {
			if (c == 0) {
				maiorProb = temp[blockDim.x + 3];
				bestClass = c;
			} 
			else if (temp[blockDim.x + 3] > maiorProb) {
				maiorProb = temp[blockDim.x + 3];
				bestClass = c;
			}
		}
		__syncthreads();
	}

	if (tid == 0) {
		probClassSp[blockIdx.x] = maiorProb;
		docClassSp[blockIdx.x] = bestClass;
	}
}


extern "C" {

double nb_gpu(const char* filenameTreino, const char* filenameTeste,
		int numDocs, int numClasses, int numTerms, int numDocsTest,
		int numTermsTest, double alpha, double lambda, int cudaDevice) {


	hipDeviceReset();
	hipSetDevice(cudaDevice);
	// clock_t begin, endT, end;
	double iTreino, fTreino;
	iTreino = get_wall_time();

	cerr << "Parametros " << alpha << " " << lambda << endl;

	int block_size, n_blocks;
	int *docTestIndexVector = (int*) malloc((numDocsTest + 1) * sizeof(int)); //Alterei numDocs para numDocsTest
	int *docTestVector = NULL;
	double *docTestFreqVector = NULL;
	int *docClassVector = (int*) malloc(numDocs * sizeof(int));

	int *freqClassVector = (int*) malloc(numClasses * sizeof(int));
	double *totalFreqClassVector = (double*) malloc(
			numClasses * sizeof(double));
	double *matrixTermFreq = (double*) malloc(
			(numTerms * numClasses) * sizeof(double));
	double *freqTermVector = (double*) malloc((numTerms) * sizeof(double));
	double totalTermFreq = 0.0;
	int totalTerms = 0;
	
	map<string, int> docAttribute;

	for (int i = 0; i < numClasses; i++) {
		totalFreqClassVector[i] = 0.0;
		freqClassVector[i] = 0;
		for (int j = 0; j < numTerms; j++) {
			matrixTermFreq[i * numTerms + j] = 0.0;
		}
	}
	for (int j = 0; j < numTerms; j++) {
		freqTermVector[j] = 0.0;
	}

	int *docIndexVector = (int*) malloc((numDocs + 1) * sizeof(int));
	int *docVector = NULL;
	double *docFreqVector = NULL;

	set<int> vocabulary;
	docVector = readTrainDataSP(filenameTreino, docIndexVector,
			totalFreqClassVector, freqClassVector, freqTermVector,
			&totalTermFreq, numClasses, numTerms, &totalTerms, matrixTermFreq,
			vocabulary, &docFreqVector, docClassVector);

	double *matrixTermFreq_D;
	hipMalloc((void **) &matrixTermFreq_D,
			sizeof(double) * (numTerms * numClasses));
	hipMemcpy(matrixTermFreq_D, matrixTermFreq,
			sizeof(double) * (numTerms * numClasses), hipMemcpyHostToDevice);


	int *freqClassVector_D;
	hipMalloc((void **) &freqClassVector_D, sizeof(int) * numClasses);
	hipMemcpy(freqClassVector_D, freqClassVector, sizeof(int) * numClasses,
			hipMemcpyHostToDevice);
	double *totalFreqClassVector_D;
	hipMalloc((void **) &totalFreqClassVector_D, sizeof(double) * numClasses);
	hipMemcpy(totalFreqClassVector_D, totalFreqClassVector,
			sizeof(double) * numClasses, hipMemcpyHostToDevice);


	double *freqTermVector_D;
	hipMalloc((void **) &freqTermVector_D, sizeof(double) * numTerms);
	hipMemcpy(freqTermVector_D, freqTermVector, sizeof(double) * numTerms,
			hipMemcpyHostToDevice);

	/* ============================ TESTE ================================*/
	int *realClass = (int*) malloc((numDocsTest + 1) * sizeof(int));

	docTestVector = readTestData(filenameTeste, docTestIndexVector, realClass,
			&docTestFreqVector, numTerms, docAttribute);

	int *docTestIndexVector_D;
	hipMalloc((void **) &docTestIndexVector_D,
			sizeof(int) * (numDocsTest + 1));
	hipMemcpy(docTestIndexVector_D, docTestIndexVector,
			sizeof(int) * (numDocsTest + 1), hipMemcpyHostToDevice);
	int *docTestVector_D;
	hipMalloc((void **) &docTestVector_D,
			sizeof(int) * docTestIndexVector[numDocsTest]);
	hipMemcpy(docTestVector_D, docTestVector,
			sizeof(int) * docTestIndexVector[numDocsTest],
			hipMemcpyHostToDevice);
	double *docTestFreqVector_D;
	hipMalloc((void **) &docTestFreqVector_D,
			sizeof(double) * docTestIndexVector[numDocsTest]);
	hipMemcpy(docTestFreqVector_D, docTestFreqVector,
			sizeof(double) * docTestIndexVector[numDocsTest],
			hipMemcpyHostToDevice);

	double *probClasse = (double*) malloc((numDocsTest) * sizeof(double));
	double *probClasse_D;
	hipMalloc((void **) &probClasse_D, sizeof(double) * (numDocsTest));

	double* modeloNB = (double*) malloc(
			(numClasses * (numDocsTest)) * sizeof(double));
	double* modeloNB_D;
	hipMalloc((void **) &modeloNB_D,
			sizeof(double) * (numClasses * (numDocsTest)));


	block_size = SIZE_CLASS;
	n_blocks = numDocsTest;
	trainning_kernel2<<<n_blocks, block_size, (block_size + 3) * sizeof(double)>>>(
			freqClassVector_D, matrixTermFreq_D, totalFreqClassVector_D,
			docTestIndexVector_D, docTestVector_D, docTestFreqVector_D,
			probClasse_D, numClasses, numTerms, numDocsTest, freqTermVector_D,
			totalTermFreq, totalTerms, lambda, alpha, numDocs, modeloNB_D);
	

	hipMemcpy(probClasse, probClasse_D, sizeof(double) * (numDocsTest),
			hipMemcpyDeviceToHost);
	hipMemcpy(modeloNB, modeloNB_D,
			sizeof(double) * (numClasses * numDocsTest),
			hipMemcpyDeviceToHost);

	double valorFinal, maiorProb;
	int maiorClasseProb;
	int *predictClass = (int*) malloc((numDocsTest) * sizeof(int));

	for (int d = 0; d < numDocsTest; d++) {
		maiorProb = modeloNB[d * numClasses + 0];
		maiorClasseProb = 0;
		for (int c = 1; c < numClasses; c++) {
			if (modeloNB[d * numClasses + c] > maiorProb) {
				maiorClasseProb = c;
				maiorProb = modeloNB[d * numClasses + c];
			}
		}
		// cerr << d << " " << maiorProb << " " << maiorClasseProb << endl;
		predictClass[d] = maiorClasseProb;
	}

	// int *correctClass = (int*) malloc(numClasses*sizeof(int));
	// for(int c = 0; c < numClasses; c++) correctClass[c] = 0;
	// cerr << "# Classes Classificadas corretamente\n";
	// for(int d = 0; d < numDocsTest; d++){
	// 	if(predictClass[d] == realClass[d]) correctClass[realClass[d]] += 1;
	// }

	// for(int c = 0; c < numClasses; c++){
	// 	cerr << c << " " << correctClass[c] << endl;
	// }

	valorFinal = evaluate(realClass, predictClass, numDocsTest, 1);
	cerr << "Resultado Naive Bayes "
			<< evaluate(realClass, predictClass, numDocsTest, 1) * 100 << " "
			<< evaluate(realClass, predictClass, numDocsTest, 0) * 100 << endl;

  	cout << "Resultado Naive Bayes "
      << evaluate(realClass, predictClass, numDocsTest, 1) * 100 << " "
      << evaluate(realClass, predictClass, numDocsTest, 0) * 100 << endl;

	hipFree(probClasse_D);


	/* ============================ SP-TAN ================================*/

	int *docIndexVector_D;
	hipMalloc((void **) &docIndexVector_D, (numDocs + 1) * sizeof(int));
	hipMemcpy(docIndexVector_D, docIndexVector, (numDocs + 1) * sizeof(int),
			hipMemcpyHostToDevice);
	int *docVector_D;
	hipMalloc((void **) &docVector_D, sizeof(int) * docIndexVector[numDocs]);
	hipMemcpy(docVector_D, docVector, sizeof(int) * docIndexVector[numDocs],
			hipMemcpyHostToDevice);
	double *docFreqVector_D;
	hipMalloc((void **) &docFreqVector_D,
			sizeof(double) * docIndexVector[numDocs]);
	hipMemcpy(docFreqVector_D, docFreqVector,
			sizeof(double) * docIndexVector[numDocs], hipMemcpyHostToDevice);
	int *docClassVector_D;
	hipMalloc((void **) &docClassVector_D, sizeof(int) * numDocs);
	hipMemcpy(docClassVector_D, docClassVector, sizeof(int) * numDocs,
			hipMemcpyHostToDevice);

	free(docIndexVector);
	free(docVector);
	free(docFreqVector);
	free(docClassVector);

	double *totalTermClassSp_D;
	hipMalloc((void **) &totalTermClassSp_D, sizeof(double) * numClasses * numTerms);
	double *probSp = (double*) malloc(numClasses * numTerms * sizeof(double));
	int sp;
	double *probSp_D;
	hipMalloc((void **) &probSp_D, sizeof(double) * numClasses * numTerms);

	double *probClassSp = (double*) malloc(numDocsTest * sizeof(double));
	double *probClassSp_D;
	hipMalloc((void **) &probClassSp_D, sizeof(double) * numDocsTest);

	int *docClassSp = (int*) malloc(numDocsTest * sizeof(int));
	int *docClassSp_D;
	hipMalloc((void **) &docClassSp_D, sizeof(int) * numDocsTest);

	double *probChildSp = (double*) malloc(
			numTerms * sizeof(double));
	double *probChildSp_D;
	hipMalloc((void **) &probChildSp_D,
			sizeof(double) * numTerms);


	block_size = 384;
	n_blocks = (numTerms + 1) / block_size
			+ ((numTerms + 1) % block_size == 0 ? 0 : 1);
	super_parent_freq<<<n_blocks, block_size>>>(docIndexVector_D, docVector_D,
			docFreqVector_D, docClassVector_D, totalTermClassSp_D, numTerms,
			numDocs, totalTerms, numClasses);

	// double *totalTermClassSp = (double*) malloc(numClasses*numTerms*sizeof(double));
	// hipMemcpy(totalTermClassSp, totalTermClassSp_D, sizeof(double)*numClasses*numTerms, hipMemcpyDeviceToHost);
	// for(int t = 0; t < 100000; t++) cerr << t << " " << totalTermClassSp[0 * numTerms + t] << " " << totalTermClassSp[1 * numTerms + t] << " " << totalTermClassSp[3 * numTerms + t] <<  endl;
	// free(totalTermClassSp);

	int *superParents = (int*) malloc((numDocsTest) * sizeof(int));
	double *probAux = (double*) malloc(numDocsTest * sizeof(double));
	for (int d = 0; d < numDocsTest; d++) {
		probAux[d] = -9999999.0;
		superParents[d] = -1;
	}

	int *hasSp_D;
	hipMalloc( (void**) &hasSp_D, sizeof(int)*numDocs);
	int * hasSp = (int*) malloc(numDocs*sizeof(int));
	// vocabulary.clear();
	// vocabulary.insert(29);
	// vocabulary.insert(50);

	// double *meanClass = (double*) malloc(numClasses*numDocsTest*sizeof(double));
	// int *denClass = (int*) malloc(numClasses*numDocsTest*sizeof(int));
	// for(int d = 0; d < numDocsTest; d++){
	// 	for(int c = 0; c < numClasses; c++){
	// 		meanClass[d*numClasses + c] = 0.0;
	// 		denClass[d*numClasses + c] = 0;
	// 	}
	// }

	// cout << "Number of attributes " << vocabulary.size() << endl;

	// clock_t b, e;
	// float time;
	// hipEvent_t start, stop;

	for (set<int>::iterator spIt = vocabulary.begin(); spIt != vocabulary.end(); ++spIt) {
		sp = *spIt;
		
		//Start timer
		double wall0 = get_wall_time();

		// hipEventCreate(&start);
		// hipEventCreate(&stop);
		// hipEventRecord(start, 0);
		block_size = SIZE_TRAIN;
		n_blocks = numDocs;
		find_sp_kernel<<<n_blocks, block_size>>>(docIndexVector_D, docVector_D, docFreqVector_D,
				numClasses, numTerms, numDocs, totalTerms, hasSp_D, sp);

		// hipMemcpy(probSp, probSp_D, sizeof(double)*numTerms*numClasses, hipMemcpyDeviceToHost);
		// for(int c = 0; c < numClasses; c++) cerr << c << " " << probSp[c * numTerms + 0] << endl;


		// if(sp == 6617){
		// 	hipMemcpy(hasSp, hasSp_D, sizeof(int)*numDocs, hipMemcpyDeviceToHost);
		// 	int cont = 0;
		// 	for(int d = 0; d < numDocs; d++){
		// 		cerr << d << " " << hasSp[d] << endl;
		// 		if(hasSp[d] == 1) cont +=1;
		// 	}
		// 	cerr << cont << endl;
		// }

		// hipEventRecord(stop, 0);
		// hipEventSynchronize(stop);
		// hipEventElapsedTime(&time, start, stop);
		// cerr << setprecision (10) << "GPU Time [ms] " << time << endl;


		// hipEventCreate(&start);
		// hipEventCreate(&stop);
		// hipEventRecord(start, 0);
		block_size = 384;
		n_blocks = (numTerms + 1) / block_size
				+ ((numTerms + 1) % block_size == 0 ? 0 : 1);
		init_tableProb<<<n_blocks, block_size>>>(numTerms, numClasses, probSp_D);

		// hipMemcpy(probSp, probSp_D, sizeof(double)*numTerms*numClasses, hipMemcpyDeviceToHost);
		// for(int t = 0; t < numTerms; t++) cerr << t << " " << probSp[0 * numTerms + t] << " " << probSp[3 * numTerms + t] << endl;

		block_size = SIZE_TRAIN;
		n_blocks = numDocs;
		compute_frequency<<<n_blocks, block_size>>>(docIndexVector_D, docVector_D, docFreqVector_D, docClassVector_D,
			numClasses, numTerms, numDocs, totalTerms, hasSp_D, sp, probSp_D);

		// hipMemcpy(probSp, probSp_D, sizeof(double)*numTerms*numClasses, hipMemcpyDeviceToHost);
		// for(int c = 0; c < numClasses; c++) cerr << c << " " << probSp[c * numTerms + 0] << endl;

		block_size = 384;
		n_blocks = (numTerms + 1) / block_size
				+ ((numTerms + 1) % block_size == 0 ? 0 : 1);
		super_parent_train<<<n_blocks, block_size>>>(docIndexVector_D,
				docVector_D, docFreqVector_D, docClassVector_D,
				totalTermClassSp_D, numTerms, numDocs, totalTerms, numClasses,
				probSp_D, sp, alpha);

		// hipEventRecord(stop, 0);
		// hipEventSynchronize(stop);
		// hipEventElapsedTime(&time, start, stop);
		// cerr << setprecision (10) << "GPU Time [ms] " << time << endl;

		// hipMemcpy(probSp, probSp_D, sizeof(double)*numTerms*numClasses, hipMemcpyDeviceToHost);
		// for(int t = 0; t < numTerms; t++){
		// 	cerr << t << " ";
			// for(int c = 0; c < numClasses; c++){
			// 	double nt = freqTermVector[t] / totalTermFreq;
			// 	double prob = (matrixTermFreq[c * numTerms + t] + alpha) / (totalFreqClassVector[c] + alpha * totalTerms);
			// 	prob = log(lambda * nt + (1.0 - lambda) * prob);  
			// 	cerr << c << " " << prob << " ";
			// 	//cerr << c << " " << probSp[c * numTerms + t] << " ";
			// }
			// cerr << endl;
		// }
		
		// hipEventCreate(&start);
		// hipEventCreate(&stop);
		// hipEventRecord(start, 0);

		block_size = SIZE_CLASS;
		n_blocks = numDocsTest;
		super_parent_predict2<<<n_blocks, block_size,
				(block_size + 3) * sizeof(double)>>>(matrixTermFreq_D,
				totalFreqClassVector_D, docTestIndexVector_D, docTestVector_D,
				docTestFreqVector_D, probClassSp_D, numClasses, numTerms,
				numDocsTest, freqTermVector_D, totalTermFreq, totalTerms,
				lambda, alpha, sp, modeloNB_D, probSp_D, docClassSp_D, freqClassVector_D, numDocs);


		// hipEventRecord(stop, 0);
		// hipEventSynchronize(stop);
		// hipEventElapsedTime(&time, start, stop);
		// cerr << setprecision (10) << "GPU Time [ms] " << time << endl;


		// b=clock();	
	
		//Avaliação da qualidade de classificação dado o Super Pai
		hipMemcpy(probClassSp, probClassSp_D, numDocsTest * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(docClassSp, docClassSp_D, numDocsTest * sizeof(int),	hipMemcpyDeviceToHost);

		// if(sp == 0){
		// 	for (int d = 0; d < numDocsTest; d++) {
		// 		cerr << d << " " << probClassSp[d] << " " << docClassSp[d] << endl;
		// 	}
		// }

		for (int d = 0; d < numDocsTest; d++) {
			// if((realClass[d] == 4 && docClassSp[d] == 4) || (realClass[d] == 9 && docClassSp[d] == 9)){
			// 	cerr << d << " " << realClass[d] << " " << sp << " " << probClassSp[d] << " " << probClasse[d]<< endl;
			// }
			// if(docAttribute[d*numTerms+sp] == 1){
			// 	meanClass[d*numClasses + docClassSp[d]] += probClassSp[d];
			// 	denClass[d*numClasses + docClassSp[d]] += 1;
			// }
			// if(docAttribute.find(index(d,sp)) != docAttribute.end()) cerr << "Doc " << d << " " <<  index(d, sp) <<  endl;
			if ((probClassSp[d] > probAux[d]) && (docAttribute.find(index(d, sp)) != docAttribute.end())){
				cerr << " ( " << d+1 << " " << probAux[d] << " -> " << probClassSp[d] << " " << docClassSp[d] << " ) " ; 
				probAux[d] = probClassSp[d];
				superParents[d] = sp;
				predictClass[d] = docClassSp[d];
			}
		}
		// e=clock();
		// cerr << "SP: " << sp << " " << evaluate(realClass, predictClass, numDocsTest, 1) * 100 << " " << evaluate(realClass, predictClass, numDocsTest, 0) * 100;
		// cerr << " Time " << double(e-b)/CLOCKS_PER_SEC;

		//Stop timers
    	double wall1 = get_wall_time();

		cerr << endl << "SP: " << sp << " " << evaluate(realClass, predictClass, numDocsTest, 1) * 100 << " " << evaluate(realClass, predictClass, numDocsTest, 0) * 100;
		cerr << " " << wall1 - wall0;
		cerr << endl;
	}
	// CUDA_SAFE_CALL(hipEventDestroy(start));
 //    CUDA_SAFE_CALL(hipEventDestroy(stop));
	free(hasSp);
	docAttribute.clear();

	// for(int d=0; d< numDocsTest; d++){
	// 	cerr << d << " " << superParents[d] << " " << probAux[d] << " " << predictClass[d] << endl;
	// }

	// cerr << "# Primeiro Teste\n";
	// for(int d = 0; d < numDocsTest; d++){
	// 	if(realClass[d] == 4 || realClass[d] == 9){
	// 		cerr << d << " ";
	// 		for(int c = 0; c < numClasses; c++){
	// 			if(denClass[d*numClasses + c] != 0)
	// 				cerr << "Classe " << c << " Numerador: " << meanClass[d*numClasses + c] << " Denominador: " << denClass[d*numClasses + c] << " Razao: " <<  (meanClass[d*numClasses + c]) / (denClass[d*numClasses + c]) << " ";
	// 		}	
	// 		cerr << endl;
	// 	}
	// }
	// free(meanClass);
	// free(denClass);

	// cerr << "# Resultado dos SP para as classes 4 e 9\n";
	// for(int d = 0; d < numDocsTest; d++){
	// 	if(realClass[d] == 4 || realClass[d] == 9)
	// 		cerr << d << " " << realClass[d] << " " << predictClass[d] << " " << superParents[d] << " " << probAux[d] << " " << probClasse[d] << endl;
	// }

	// cerr << "TESTE DE SANIDADE\n";
	// for(int d =0; d < numDocsTest; d++){
	// 	cerr << d << " " << predictClass[d] << " " << probAux[d] << endl;
	// }

	// for(int c = 0; c < numClasses; c++) correctClass[c] = 0;
	// cerr << "# Classes Classificadas corretamente\n";
	// for(int d = 0; d < numDocsTest; d++){
	// 	if(predictClass[d] == realClass[d]) correctClass[realClass[d]] += 1;
	// }

	// for(int c = 0; c < numClasses; c++){
	// 	cerr << c << " " << correctClass[c] << endl;
	// }
	// free(correctClass);
	// endT = clock();
	cerr << "Melhor SP " << evaluate(realClass, predictClass, numDocsTest, 1) * 100 << " "
			<< evaluate(realClass, predictClass, numDocsTest, 0) * 100 << endl;
	cout << "Melhor SP " << evaluate(realClass, predictClass, numDocsTest, 1) * 100 << " "
			<< evaluate(realClass, predictClass, numDocsTest, 0) * 100 << endl;


	// ofstream predict("predict.dat");
	// for(int d=0; d < numDocsTest; d++){
	// 	predict << d << " real " << realClass[d] << " predict " << predictClass[d] << endl;
	// }
	// predict.close();
	
	hipFree(docIndexVector_D);
	hipFree(docVector_D);
	hipFree(docFreqVector_D);
	hipFree(docClassVector_D);
	hipFree(docTestIndexVector_D);
	hipFree(docTestVector_D);
	hipFree(docTestFreqVector_D);
	hipFree(freqTermVector_D);
	free(freqTermVector);

	free(docClassSp);
	hipFree(docClassSp_D);
	hipFree(totalFreqClassVector_D);
	free(totalFreqClassVector);

	hipFree(matrixTermFreq_D);
	free(matrixTermFreq);

	hipFree(hasSp_D);
	hipFree(totalTermClassSp_D);

	hipFree(probClassSp_D);
	free(probClassSp);

	hipFree(modeloNB_D);
	free(modeloNB);

	hipFree(probSp_D);
	free(probSp);

	free(realClass);
	free(predictClass);
	free(probAux);
	free(superParents);
	free(docTestIndexVector);
	free(docTestVector);
	free(docTestFreqVector);

	hipFree(probChildSp_D);
	free(probChildSp);
	// end = clock();
	fTreino = get_wall_time();
	cerr << "Time " <<  fTreino - iTreino << endl;
  	cout << "Time " <<  fTreino - iTreino << endl;

	return valorFinal;
}
}
